#include "hip/hip_runtime.h"
#include "caffe/layers/rcnn_output_layer.hpp"
#include "caffe/layers/proposal_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/nms.hpp"

namespace caffe {

template <typename Dtype>
__global__ static
void transform_inv_crop(const int num_box, const int num_class,
                        Dtype* bboxes, const Dtype* rois,
                        const Dtype max_w, const Dtype max_h)
{
  bboxes += (blockIdx.x * blockDim.x + threadIdx.x) * 5;
  rois += threadIdx.x * 5;
  // corner to center
  Dtype w = rois[3] - rois[1] + 1;
  Dtype h = rois[4] - rois[2] + 1;
  Dtype cx = rois[1] + 0.5 * w;
  Dtype cy = rois[2] + 0.5 * h;
  // transform back
  Dtype pcx = bboxes[0] * w + cx;
  Dtype pcy = bboxes[1] * h + cy;
  Dtype pw = exp(bboxes[2]) * w;
  Dtype ph = exp(bboxes[3]) * h;
  // center to corner
  bboxes[0] = pcx - 0.5 * pw;
  bboxes[1] = pcy - 0.5 * ph;
  bboxes[2] = pcx + 0.5 * pw;
  bboxes[3] = pcy + 0.5 * ph;
  // clip bbox to be within the image region
  bboxes[0] = max((Dtype)0,  min(bboxes[0],  max_w));
  bboxes[1] = max((Dtype)0,  min(bboxes[1],  max_h));
  bboxes[2] = max((Dtype)0,  min(bboxes[2],  max_w));
  bboxes[3] = max((Dtype)0,  min(bboxes[3],  max_h));
}

template <typename Dtype>
void bbox_filter(Dtype* bboxes, int& num,
                 const int num_box, const Dtype thresh)
{
  for (int i = 0; i < num_box; i++) {
    if (bboxes[i*5+4] >= thresh) {
      if (i > num) {
        bboxes[num*5]   = bboxes[i*5];  // x1
        bboxes[num*5+1] = bboxes[i*5+1];  // y1
        bboxes[num*5+2] = bboxes[i*5+2];  // x2
        bboxes[num*5+3] = bboxes[i*5+3];  // y2
        bboxes[num*5+4] = bboxes[i*5+4];  // score
      }
      num++;
    }
  }
}

// copy box_pred to bboxes
// layout: [n, c, 4] => [c, n, 5]
template <typename Dtype>
__global__ static
void box_transpose_copy(const int num_box, const int num_class,
                        Dtype* bboxes, const Dtype* box_pred)
{
  bboxes += (blockIdx.x * num_box + threadIdx.x) * 5;
  box_pred += (threadIdx.x * num_class + blockIdx.x) * 4;
  bboxes[0] = box_pred[0]; // x1
  bboxes[1] = box_pred[1]; // y1
  bboxes[2] = box_pred[2]; // x2
  bboxes[3] = box_pred[3]; // y2
}

template <typename Dtype>
__global__ static
void cls_transpose_copy(const int num_box, const int num_class,
                        Dtype* bboxes, const Dtype* cls_pred)
{
  bboxes[(blockIdx.x * num_box + threadIdx.x) * 5 + 4] 
      = cls_pred[threadIdx.x * num_class + blockIdx.x];
}

template <typename Dtype>
__global__ static
void all_result_copy(const int num_result, const int num_box,
                     Dtype* result, const Dtype* bboxes, const int* cood)
{
  cood += threadIdx.x * 2;
  bboxes += (cood[0] * num_box + cood[1]) * 5;
  result += threadIdx.x * 6;
  result[0] = cood[0] - 1;  // foreground class id
  result[1] = bboxes[4];  // score
  result[2] = bboxes[0];  // x1
  result[3] = bboxes[1];  // y1
  result[4] = bboxes[2];  // x2
  result[5] = bboxes[3];  // y2
}

template <typename Dtype>
void RCNNOutputLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top)
{
  // prepare a dummy output as pycaffe doesn't support empty data export.
  top[0]->Reshape({1});
  top[0]->mutable_cpu_data()[0] = 0;
  
  // case: empty input (no proposals) 
  if (bottom[0]->count() == 0)
    return;
  
  // bottoms: box_pred, cls_pred, rois, im_info
  const Dtype* box_pred = bottom[0]->gpu_data();
  const Dtype* cls_pred = bottom[1]->gpu_data();
  const Dtype* rois = bottom[2]->gpu_data();
  const Dtype* info = bottom[3]->cpu_data();

  const int num_box = bottom[0]->shape(0);
  const int num_class = bottom[1]->shape(1);
  const Dtype max_h = info[0] - 1;
  const Dtype max_w = info[1] - 1;
  const Dtype scale = 1. / info[2];

  // (c, n, 5) [score, x1, y1, x2, y2]
  Blob<Dtype> bboxes({num_class, num_box, 5});
  box_transpose_copy<Dtype><<<num_class, num_box>>>(
      num_box, num_class, bboxes.mutable_gpu_data(), box_pred);
  transform_inv_crop<Dtype><<<num_class, num_box>>>(
      num_box, num_class, bboxes.mutable_gpu_data(), rois, max_w, max_h);
  caffe_gpu_scal(bboxes.count(), scale, bboxes.mutable_gpu_data());
  cls_transpose_copy<Dtype><<<num_class, num_box>>>(
      num_box, num_class, bboxes.mutable_gpu_data(), cls_pred);

  // nms on every foreground class
  int num_result = 0;
  Blob<int> coords({num_box * num_class, 2});
  int* cood = coords.mutable_cpu_data();
  Blob<int> indices({num_box});
  Blob<int> mask;
  for (int c = 1; c < num_class; c++) {
    int num = 0;
    Dtype* bdata = bboxes.mutable_cpu_data() + c * num_box * 5;
    bbox_filter(bdata, num, num_box, conf_thresh_);
    sort_box(bdata, 0, num-1, num);
    if (num == 0)
      continue;
    nms_gpu(num, bboxes.gpu_data() + c * num_box * 5, &mask, 
            indices.mutable_cpu_data(), &num, 0, nms_thresh_, -1);
    for (int i = 0; i < num; i++) {
      cood[0] = c;
      cood[1] = indices.cpu_data()[i];
      cood += 2;
    }
    num_result += num;
  }
  if (num_result == 0)
    return;
  
  // copy all result
  CHECK_GT(num_result, 0);
  top[0]->Reshape({num_result, 6});
  all_result_copy<Dtype><<<1, num_result>>>(
      num_result, num_box, top[0]->mutable_gpu_data(),
      bboxes.gpu_data(), coords. gpu_data()
  );
}

template <typename Dtype>
void RCNNOutputLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  NOT_IMPLEMENTED;
}

INSTANTIATE_LAYER_GPU_FUNCS(RCNNOutputLayer);


}  // namespace caffe
